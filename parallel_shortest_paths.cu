#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cmath>
#include <map>
#include "graph.cpp"


using namespace std;

void print(vector<int> &v){
    for(int i=0;i<v.size();i++){
        cout<<v[i]<<" ";
    }
    cout<<endl;
}
__global__ void delta_step_SSSP(Graph graph){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < graph.num_vertices){
        int u = tid;
        for(int i=graph.vertices[u];i<graph.vertices[u+1];i++){
            int v = graph.edges[i];
            int w = graph.weights[i];
            if(graph.distances[v] > graph.distances[u] + w){
                graph.distances[v] = graph.distances[u] + w;
            }
        }
    }
}

__global__ 
void vertex_out_degree(Graph graph){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < graph.edges; i++)
    {
        if (tid == graph.edges[i][0])
        {
            graph.degree[tid]++;
        }
    }
}
void degrees_parallel(Graph graph){
    const size_t THREADS_PER_BLOCK = 256;
    hipMallocManaged(&graph.degree, vertices * sizeof(int));
    for (int i = 0; i < vertices; i++)
    {
        graph.degree[i] = 0;
    }
    const size_t block_size = (vertices + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vertex_out_degree<<<block_size, THREADS_PER_BLOCK>>>(graph);
    hipDeviceSynchronize();
    vector<int> degrees;
    for (int i = 0; i < vertices; i++)
    {
        degrees.push_back(graph.degree[i]);
    }
    //write to graph degrees
    graph.setDegrees(degrees);
}

int max_degree_parallel(Graph graph){
    graph.degrees_parallel();
    int *max_degree = max_element(graph.degree, graph.degree + graph.getN());
    return *max_degree;
}

void delta_step_SSSP(Graph graph){
    int max_degree = max_degree_parallel(graph);
    
}
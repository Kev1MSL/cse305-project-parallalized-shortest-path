#include "hip/hip_runtime.h"
#include graph.h

using namespace std;

void print(vector<int> &v){
    for(int i=0;i<v.size();i++){
        cout<<v[i]<<" ";
    }
    cout<<endl;
}
__global__ void delta_step_SSSP(Graph graph){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < graph.num_vertices){
        int u = tid;
        for(int i=graph.vertices[u];i<graph.vertices[u+1];i++){
            int v = graph.edges[i];
            int w = graph.weights[i];
            if(graph.distances[v] > graph.distances[u] + w){
                graph.distances[v] = graph.distances[u] + w;
            }
        }
    }
}

__global__ 
void vertex_out_degree(Graph graph){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < graph.edges; i++)
    {
        if (tid == graph.edges[i][0])
        {
            graph.degree[tid]++;
        }
    }

}

// 
//TODO: test max degree parallel
int max_degree(Graph graph){
    // assuming that degrees are already found.

    int *max_degree = max_element(graph.degree, graph.degree + graph.getN());
    return *max_degree;
}

/**
 * @brief SSSP using delta stepping algorithm
 * @param graph a graph object with N vertices and M edges
 */
void delta_step_SSSP(Graph graph){
    int max_degree = max_degree(graph); //maxdegree to find optimal delta
}
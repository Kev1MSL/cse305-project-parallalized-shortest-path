#include "hip/hip_runtime.h"
#include "graph.h"
#include <algorithm>
#include <set>
#include <numeric>
#include <cmath>
using namespace std;
using Request_set = std::set<std::tuple<int, int, int>>; //define alias type for request


// __device__ void relax(int u, int v, double w, double* distances_d, double delta, int* p, set<int>* buckets){ 
//     //compute distance to v via u vs distance to v already. (this function assumes (u,v, w) not (v, u, w)))
//     double tentative = distances_d[u] + w;
//     if (distances_d[v] > tentative){
//         distances_d[v] = tentative;
//         int i = floor(distances_d[v]/delta);
//         int j = floor(tentative/delta);
//         buckets[i].erase(v);
//         buckets[j].insert(v); //move edge to new bucket 
//         p[u] = 1;

//     }
// }

// loop 3
// __global__ void relaxRequestsParallel(Request_set requests, double* distances_d, double delta, int* p, set<int>* buckets){
//     int tid = threadIdx.x + blockIdx.x * blockDim.x;
//     for (auto request : requests){
//         requests.erase(request);
//         relax(std::get<0>(request), std::get<1>(request), std::get<2>(request), distances_d, delta, p, buckets);
//     } 
// }
// 
//TODO: test max degree parallel
int max_degree(Graph graph){
    // assuming that degrees are already found.
    int max_degree = *std::max_element(graph.getGraphDegrees().begin(), graph.getGraphDegrees().end());
    return max_degree;
}


/**
 * @brief find an generate requests chunk by chunk on gpu approach
    * @param graph a graph object with N vertices and M edges
    * @param k delta bucket index
    * @param chunk_size the size of the chunk to be processed by each thread
    * @param N_light the light neighbours of each vertex, list of dynamic length vectors
    * @param N_heavy the heavy neighbours of each vertex, list of dynamic length vectors
    * @param delta the delta value
 */
// __global__ void
// findRequestsParallel(Request_set* request_heavy,
//                 Request_set* request_light,
//                 Graph* graph, 
//                 int k, 
//                 size_t chunk_size,
//                 vector* N_light,
//                 vector* N_heavy,
//                 double delta){
//     int tid = threadIdx.x + blockIdx.x * blockDim.x;
//     size_t begin = tid * chunk_size;
//     size_t end = begin + chunk_size;
//     if (end > graph.getGraphNbVertices()){
//         end = graph.getGraphNbVertices();
//     }
//     for (int v = begin; v < end; v++){
//         //GenRequests 
//         for (size_t i = 0; i < N_light[v].size(); i++){
//             request_light.insert(std::make_tuple(v, u, graph.getEdgeWeight(v, u)));
//         }
//         for (size_t i = 0; i < N_heavy[v].size(); i++){
//             request_heavy.insert(std::make_tuple(v, u, graph.getEdgeWeight(v, u)));
//         }
//     }
   
// }

__global__ void lightHeavyNeighborsParallel(double** AdjMat, size_t N, std::vector<int>* N_light, std::vector<int>* N_heavy, double delta, size_t chunk_size){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    size_t begin = tid * chunk_size;
    size_t end = begin + chunk_size;
    if (end > N){
        end = N;
    }
    
    //use adjacency matrix to find light and heavy neighbours
    for (int v = begin; v < end; v++){
        for (int u = 0; u < N; u++){
            if (AdjMat[v][u] <= delta){
                N_light[v].push_back(u);
            }
            else{
                N_heavy[v].push_back(u);
            }
        }
    }
}
void delta_step_SSSP_Parallel(Graph graph, int start = 0){
    const size_t THREADS_PER_BLOCK = 96;
    int bucket_index = 0;
    const size_t NUM_THREADS = max_degree(graph); //maxdegree to find optimal delta
    double delta = 1/NUM_THREADS;
    size_t chunk_size = graph.getGraphNbVertices() / NUM_THREADS;
    const size_t NUM_BLOCKS = (NUM_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    std::vector<int>* N_light;
    std::vector<int>* N_heavy;
    hipMalloc(&N_light, graph.getGraphNbVertices() * sizeof(vector<int>));
    hipMalloc(&N_heavy, graph.getGraphNbVertices() * sizeof(vector<int>));
    //copy adj matrix to device
    std::vector<std::vector<double>> AdjMat = graph.getAdjMatrix();
    double** AdjMat_d; 
    for (int i = 0; i < graph.getGraphNbVertices(); i++){
        hipMalloc(&AdjMat_d[i], graph.getGraphNbVertices() * sizeof(double));
        hipMemcpy(AdjMat_d[i], AdjMat[i].data(), graph.getGraphNbVertices() * sizeof(double), hipMemcpyHostToDevice);
    }

    //find light and heavy neighbours
    lightHeavyNeighborsParallel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(AdjMat_d, graph.getGraphNbVertices(), N_light, N_heavy, delta, chunk_size);
    hipDeviceSynchronize();

    std::vector<int>* N_light_h = (std::vector<int>*)malloc(graph.getGraphNbVertices() * sizeof(vector<int>));
    std::vector<int>* N_heavy_h = (std::vector<int>*)malloc(graph.getGraphNbVertices() * sizeof(vector<int>));
    hipMemcpy(N_light_h, N_light, graph.getGraphNbVertices() * sizeof(vector<int>), hipMemcpyDeviceToHost);
    hipMemcpy(N_heavy_h, N_heavy, graph.getGraphNbVertices() * sizeof(vector<int>), hipMemcpyDeviceToHost);
    hipFree(N_light);
    hipFree(N_heavy);
    //print result 
    for(int i=0;i<graph.getGraphNbVertices();i++){
        cout<<"N_light["<<i<<"] = ";
        for(int j=0;j<N_light_h[i].size();j++){
            cout<<N_light_h[i][j]<<" ";
        }
        cout<<"N_heavy["<<i<<"] = ";
        for(int j=0;j<N_heavy_h[i].size();j++){
            cout<<N_heavy_h[i][j]<<" ";
        }
       
    }
}
/**
 * @brief SSSP using delta stepping algorithm
 * @param graph a graph object with N vertices and M edges
 */
// void delta_step_SSSP_Parallel(Graph graph, int start = 0){
//     const size_t THREADS_PER_BLOCK = 96;
//     int bucket_index = 0;
//     const size_t NUM_THREADS = max_degree(graph); //maxdegree to find optimal delta
//     const size_t NUM_BLOCKS = (NUM_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

//     //initialize distances output and also distances on device
//     // vector<double> distances(graph.getGraphNbVertices(), std::numeric_limits<double>::infinity());
//     // distances[start] = 0;
//     // double* distances_d;
//     // hipMalloc(&distances_d, graph.getGraphNbVertices() * sizeof(double));
//     // hipMemcpy(distances_d, distances.data(), graph.getGraphNbVertices() * sizeof(double), hipMemcpyHostToDevice);

//     // //initialize buckets
//     // //add all vertices to bucket_inf except start
        set<int> bucket_inf;
        set::iota(std::inserter(bucket_inf, bucket_inf.begin()), 0, graph.getGraphNbVertices());
//     // bucket_inf.erase(start);
//     // //as all the weights between 0 and 1, the max path distance should be less than the number of vertices
//     // vector<set<int>> buckets[graph.getGraphNbVertices()]; 
//     // buckets[0].insert(start);

//     // int k = 0;
//     //test light and heavy neighbours parallel:
//     std::vector* N_light;
//     std::vector* N_heavy;
//     hipMalloc(&N_light, graph.getGraphNbVertices() * sizeof(vector));
//     hipMalloc(&N_heavy, graph.getGraphNbVertices() * sizeof(vector));
//     lightHeavyNeighborsParallel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(graph, N_light, N_heavy, delta, chunk_size);
//     hipDeviceSynchronize();
//     //print result 
//     for(int i=0;i<graph.getGraphNbVertices();i++){
//         cout<<"N_light["<<i<<"] = ";
//         printVector<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(N_light[i]);
//         cout<<"N_heavy["<<i<<"] = ";
//         printVector<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(N_heavy[i]);
//     }
//     // while (k < std::numeric_limits<double>::infinity()){
//     //     Request_set request_heavy, request_light; // (u, v, w) tuples
//     //     while (!buckets[k].empty()){
            
//     //     }
//     // }
// }
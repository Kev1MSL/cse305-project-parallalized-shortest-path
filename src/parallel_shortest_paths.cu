#include "hip/hip_runtime.h"
#include graph.h

using namespace std;

void print(vector<int> &v){
    for(int i=0;i<v.size();i++){
        cout<<v[i]<<" ";
    }
    cout<<endl;
}
__global__ void delta_step_SSSP(Graph graph){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < graph.num_vertices){
        int u = tid;
        for(int i=graph.vertices[u];i<graph.vertices[u+1];i++){
            int v = graph.edges[i];
            int w = graph.weights[i];
            if(graph.distances[v] > graph.distances[u] + w){
                graph.distances[v] = graph.distances[u] + w;
            }
        }
    }
}


// 
//TODO: test max degree parallel
int max_degree(Graph graph){
    // assuming that degrees are already found.

    int *max_degree = max_element(graph.degree, graph.degree + graph.getN());
    return *max_degree;
}

/**
 * @brief SSSP using delta stepping algorithm
 * @param graph a graph object with N vertices and M edges
 */
void delta_step_SSSP(Graph graph, int start = 0){
    const size_t THREADS_PER_BLOCK = 96;
    int bucket_index = 0;
    const size_t NUM_THREADS = max_degree(graph); //maxdegree to find optimal delta
    const size_t NUM_BLOCKS = (NUM_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    //initialize distances to infinity
    double distances = new double[graph.getN()];
    for(int i=0;i<graph.getN();i++){
        distances[i] = numeric_limits<double>::max();
    }
    distances[start] = 0;
    //copy distances to tentative
    double *tentative_distances = new double[graph.getN()];
    for(int i=0;i<graph.getN();i++){
        tentative_distances[i] = distances[i];
    }
    
    //copy tentative distances to device
    double *d_tentative_distances;
    hipMalloc(&d_tentative_distances, graph.getN() * sizeof(double));
    hipMemcpy(d_tentative_distances, tentative_distances, graph.getN() * sizeof(double), hipMemcpyHostToDevice);

    while(true){
        
    }

}